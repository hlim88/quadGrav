/* Matt Anderson
 * 21 Mar 2009
 * Neutrino transport via Monte Carlo on the GPU
 */



#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <cutil.h>
#include "MersenneTwister.h"

////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}

float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}


///////////////////////////////////////////////////////////////////////////////
// GPU code
///////////////////////////////////////////////////////////////////////////////
#include "MersenneTwister_kernel.cu"
#include "Neutrino_kernel.cu"



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////
//Simulation paths (random samples) count 
//const int PATH_N = 24000000;
//Number of outputs per generator; align to even for Box-Muller transform
//const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
//Total numbers of sample to generate
//const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

//Reduce problem size to have reasonable emulation time
const int  OPT_N = 1;
//#ifndef __DEVICE_EMULATION__
//const int  OPT_N = 128;
//#else
//const int  OPT_N = 4;
//#endif


///////////////////////////////////////////////////////////////////////////////
// Start off point for neutrino transport calculation
///////////////////////////////////////////////////////////////////////////////
extern "C" void neutrino_(double *etemp,double *edens,double *xx,double *yy, double *zz)
{
    float
        *d_Random;

    float
        h_particlesGPU[OPT_N],
        h_r2barsqGPU[OPT_N],
        x[OPT_N];

    double gpuTime;

    //Simulation paths (random samples) count 
    int PATH_N = 10000;
    //Number of outputs per generator; align to even for Box-Muller transform
    int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 12);
    //Total numbers of sample to generate
    int RAND_N = MT_RNG_COUNT * N_PER_RNG;
    
    // the start particle energy -- eV
    double E0 = 0.0;

    // the cutoff particle energy -- eV
    double Ef = 1.4;

    // atomic density of hydrogen and oxygen: units g/cm^3
    double hdens, odens;
    hdens = 0.06692;
    odens = 0.03346;

    // radius of fluid
    double radius = 300.0;

    // random seed
    int iseed = 5;

    unsigned int hTimer;

    CUT_DEVICE_INIT();
    CUT_SAFE_CALL( cutCreateTimer(&hTimer) );

    printf("Loading GPU twisters configurations...\n");
    
    //const char *dat_path = cutFindFilePath("MersenneTwister.dat", argv[0]);
    //  initMTGPU(dat_path);
      initMTGPU("/home/matt/had_cvs/had/src/hyperGHMHD/MersenneTwister.dat");

    printf("Generating random options...\n");
    
    CUDA_SAFE_CALL( hipMalloc((void **)&d_Random, 12*RAND_N  * sizeof(float)) );

    srand(iseed);

    printf("Data init done.\n");

    printf("RandomGPU()...\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    RandomGPU<<<32, 128>>>(d_Random, N_PER_RNG, 777);
    CUT_CHECK_ERROR("RandomGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime = cutGetTimerValue(hTimer);
    printf("Generated samples : %i \n", RAND_N);
    printf("RandomGPU() time  : %f ms\n", gpuTime);
    printf("Samples per second: %E \n", RAND_N / (gpuTime * 0.001));

    printf("Scattering...\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    BoxMullerGPU<<<32, 128>>>(d_Random, N_PER_RNG,
                              E0,Ef,hdens,odens,radius);
    CUT_CHECK_ERROR("ScatteringGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime = cutGetTimerValue(hTimer);
    printf("Scattering time : %f ms\n", gpuTime);
    printf("Samples per second  : %E \n", RAND_N / (gpuTime * 0.001));


    printf("GPU Monte-Carlo simulation...\n");
    initMonteCarloGPU(OPT_N, PATH_N,x);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    MonteCarloGPU(
             h_r2barsqGPU, h_particlesGPU,
             OPT_N, d_Random, PATH_N);

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    closeMonteCarloGPU();
    gpuTime = cutGetTimerValue(hTimer);
    printf("Total GPU time     : %f ms\n", gpuTime);
    printf("Neutron lifetime: : %f %f \n",h_r2barsqGPU[0]/6.0,h_particlesGPU[0]);
    
    printf("Shutting down...\n");
    CUDA_SAFE_CALL( hipFree(d_Random) );

    CUT_SAFE_CALL( cutDeleteTimer( hTimer) );

}
