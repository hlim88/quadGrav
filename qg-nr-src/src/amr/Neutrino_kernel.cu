#include "hip/hip_runtime.h"
/* 21 Mar 2009
 * Matt Anderson
 * Code for neutrino transport
 */
static const int MAX_OPTIONS = 256;
static const int THREAD_N = 256;

// Determined empirically for G80 GPUs
// Experiment with this value to obtain higher performance on 
// GPUs with more or fewer multiprocessors
static const int MULTIBLOCK_THRESHOLD = 8192;

static float *d_Sum;
static float *h_Sum;

__constant__ float d_X[MAX_OPTIONS];

////////////////////////////////////////////////////////////////////////////////////
// Compute an efficient number of CTAs to use per option for the multiblock
// version (MonteCarloKernel()).  These numbers were determined via experimentation
// on G80 GPUs.  Optimal values may be different on other GPUs.
////////////////////////////////////////////////////////////////////////////////////
unsigned int computeNumCTAs(unsigned int optN)
{
    return (optN < 16) ? 64 : 16;
}

////////////////////////////////////////////////////////////////////////////////////
// Allocate intermediate strage for the Monte Carlo integration
////////////////////////////////////////////////////////////////////////////////////
void initMonteCarloGPU(unsigned int optN, unsigned int pathN, float *h_X){
    
    unsigned int ratio = pathN / optN;
    unsigned int accumSz = 2 * sizeof(float);
    if (ratio >= MULTIBLOCK_THRESHOLD) 
    {
        // in this case we need to store a number of partial sums per thread block
        unsigned int accumN = computeNumCTAs(optN) * THREAD_N;
        accumSz *= accumN;
    }

    ////////////////////////////////////////////////////////////////////////////
    // Use OS-pinned memory on host side. Allocation takes slightly more time,
    // But OS-pinned<==>device memory transfers are faster depending on 
    // the system configuration. Refer to the programming guide and 
    // bandwidthTest CUDA SDK sample for performance comparisons on the
    // particular system.
    ////////////////////////////////////////////////////////////////////////////
    CUDA_SAFE_CALL( hipHostMalloc((void **)&h_Sum,  optN * 2 * sizeof(float)) );
    CUDA_SAFE_CALL( hipMalloc((void **)&d_Sum,  accumSz * optN) );

    // Initialize the per-option data in constant arrays accessible by MonteCarloKernel()
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(d_X), h_X, MAX_OPTIONS * sizeof(float)) );
}

void closeMonteCarloGPU(void){
    CUDA_SAFE_CALL( hipFree(d_Sum)      );
    CUDA_SAFE_CALL( hipHostFree(h_Sum)  );
}

// Needed by the optimized sum reduction for correct execution in device emulation
#ifdef __DEVICE_EMULATION__
#define SYNC __syncthreads()
#else
#define SYNC
#endif

////////////////////////////////////////////////////////////////////////////////////
// Given shared memory with blockSize valus and blockSize squared values,
// This function computes the sum of each array.  The result for each array
// is stored in element 0 of tha array.
////////////////////////////////////////////////////////////////////////////////////
template <unsigned int blockSize>
__device__ void 
sumReduceSharedMem(float *sum, float *sum2)
{
    unsigned int tid = threadIdx.x;

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sum[tid] += sum[tid + 256]; sum2[tid] += sum2[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sum[tid] += sum[tid + 128]; sum2[tid] += sum2[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sum[tid] += sum[tid +  64]; sum2[tid] += sum2[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { sum[tid] += sum[tid + 32]; sum2[tid] += sum2[tid + 32]; SYNC; }
        if (blockSize >=  32) { sum[tid] += sum[tid + 16]; sum2[tid] += sum2[tid + 16]; SYNC; }
        if (blockSize >=  16) { sum[tid] += sum[tid +  8]; sum2[tid] += sum2[tid +  8]; SYNC; }
        if (blockSize >=   8) { sum[tid] += sum[tid +  4]; sum2[tid] += sum2[tid +  4]; SYNC; }
        if (blockSize >=   4) { sum[tid] += sum[tid +  2]; sum2[tid] += sum2[tid +  2]; SYNC; }
        if (blockSize >=   2) { sum[tid] += sum[tid +  1]; sum2[tid] += sum2[tid +  1]; SYNC; }
    }
}

////////////////////////////////////////////////////////////////////////////////////
// Compute the final sum and sum-of-squares of ACCUM_N values for each option using 
// an optimized parallel tree reduction.  Calls sumReduceSharedMem
////////////////////////////////////////////////////////////////////////////////////
template <unsigned int blockSize>
__global__ void
sumReduction(float *g_odata, float *g_idata, unsigned int blockDataSize)
{
    __shared__ float sum[blockSize];
    __shared__ float sum2[blockSize]; // sum of squares

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*2*blockDataSize + threadIdx.x;
    sum[tid] = 0;
    sum2[tid] = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridSize).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    for (int count = 0; count < blockDataSize/blockSize; count++)
    {
        sum[tid]  += g_idata[i];  
        sum2[tid] += g_idata[i + blockDataSize];
        i += blockSize;        
    } 
    __syncthreads();

    // do reduction in shared mem
    sumReduceSharedMem<blockSize>(sum, sum2);
    
    // write result for this block to global mem 
    if (tid == 0) 
    {
        g_odata[2 * blockIdx.x]     =  sum[0];
        g_odata[2 * blockIdx.x + 1] = sum2[0];

    }
}

////////////////////////////////////////////////////////////////////////////////////
// This kernel computes partial integrals over all paths using a multiple thread 
// blocks per option.  It is used when a single thread block per option would not
// be enough to keep the GPU busy.  Execution of this kernel is followed by
// a sumReduction() to get the complete integral for each option.
////////////////////////////////////////////////////////////////////////////////////
__global__ void MonteCarloKernel(
    float *d_Sum,    //Partial sums (+sum of squares) destination
    int   accumN,    //Partial sums (sum of squares) count
    float *d_Random, //N(0, 1) random samples array
    int   pathN
){
    const int tid      = blockDim.x * blockIdx.x + threadIdx.x;
    const int optIndex = blockIdx.y;
    const int threadN  = blockDim.x * gridDim.x;
    float r2;

    for(int iAccum = tid; iAccum < accumN; iAccum += threadN) {
      float sum = 0, sum2 = 0;
      for (int iPath = iAccum; iPath < pathN; iPath += accumN) {
        r2 = d_Random[iPath];
        if ( r2 > 0.0 ) {
          sum += r2;
          sum2 += 1.0;
        }
      }
      d_Sum[optIndex * 2 * accumN + iAccum +      0] = sum;
      d_Sum[optIndex * 2 * accumN + iAccum + accumN] = sum2;
    }


  //  for(int iAccum = tid; iAccum < accumN; iAccum += threadN) {
  //    float sum = 0, sum2 = 0;
  //    for (int iPath = iAccum; iPath < pathN; iPath += accumN) {
  //      myrand = d_Random[iPath];
  //
  //       sum += myrand;
  //      sum2 += 1.0;
  //    }
  //    d_Sum[optIndex * 2 * accumN + iAccum +      0] = sum;
  //    d_Sum[optIndex * 2 * accumN + iAccum + accumN] = sum2;
  //  }
}

////////////////////////////////////////////////////////////////////////////////////
// Here we choose between two different methods for performing Monte Carlo 
// integration on the GPU.  When the ratio of paths to options is lower than a 
// threshold (8192 determined empirically for G80 GPUs -- a different threshold is 
// likely to be better on other GPUs!), we run a single kernel that runs one thread 
// block per option and integrates all samples for that option.  This is 
// MonteCarloKernelOneBlockPerOption().  When the ratio is high, then we need more
// threads to better hide memory latency.  In this case we run multiple thread
// blocks per option and compute partial sums stored in the d_Sum array.  This is
// MonteCarloKernel().  These partial sums are then reduced to a final sum using  
// a parallel reduction (sumReduction()).  In both cases, the sum and sum of 
// squares for each option is read back to the host where the final callResult and
// confidenceWidth are computed.  These are computed on the CPU because doing so on
// the GPU would leave most threads idle.
////////////////////////////////////////////////////////////////////////////////////
void MonteCarloGPU(
    float *r2barsq,      // r^2 bar squared
    float *particles,    // number of particles
    int optN,               //Input options count
    float *d_Random,        //(0, 1) random samples array
    int pathN
){

    int ctaN = computeNumCTAs(optN);
    int accumN = ctaN * THREAD_N;
    dim3 gridDim(ctaN, optN, 1);
    MonteCarloKernel<<<gridDim, THREAD_N>>>(
        d_Sum, accumN, d_Random, pathN);
    CUT_CHECK_ERROR("MonteCarloKernel() execution failed\n");
       
    // Perform a parallel sum reduction on the device to reduce the ACCUM_N values
    // generated per option to a single value (actually two values -- sum and sum of 
    // squares).  This reduction is very efficient on the GPU.
    sumReduction<128><<<optN, 128>>>(d_Sum, d_Sum, accumN);
    CUT_CHECK_ERROR("sumReduction() execution failed\n");
    
    // Read back only the sum and sum of squares for each option to the CPU.
    CUDA_SAFE_CALL( hipMemcpy(h_Sum, d_Sum, 2 * optN * sizeof(float), hipMemcpyDeviceToHost) );

    // Compute final statistics
    for(int opt = 0; opt < optN; opt++){
        float sum  = h_Sum[2*opt];
        float sum2 = h_Sum[2*opt+1];

        //Derive average from the total sum and discount by riskfree rate 
        r2barsq[opt] = (float)( sum / sum2);
        //r2barsq[opt] = (float)( sum );
        //waveResult[opt] = (float)( sum );

        //Standard deviation
        //double stdDev = sqrt(((double)pathN * sum2 - sum * sum)/ ((double)pathN * (double)(pathN - 1)));

        //Confidence width; in 95% of all cases theoretical value lies within these borders
        //confidenceWidth[opt] = (float)(1.96 * stdDev / sqrt((double)pathN));
        particles[opt] = (float)(sum2);
    }
}
