#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* 15 Apr 2009 
 * Matt Anderson
 * Code to do neutron scattering for tests
 */

#include "MersenneTwister.h"



__device__ static mt_struct_stripped ds_MT[MT_RNG_COUNT];
static mt_struct_stripped h_MT[MT_RNG_COUNT];



void initMTGPU(const char *fname){
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        printf("initMTGPU(): failed to open %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        printf("initMTGPU(): failed to load %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    fclose(fd);

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), h_MT, sizeof(h_MT)) );
}



////////////////////////////////////////////////////////////////////////////////
// Write MT_RNG_COUNT vertical lanes of NPerRng random numbers to *d_Random.
// For coalesced global writes MT_RNG_COUNT should be a multiple of warp size.
// Initial states for each generator are the same, since the states are
// initialized from the global seed. In order to improve distribution properties
// on small NPerRng supply dedicated (local) seed to each twister.
// The local seeds, in their turn, can be extracted from global seed
// by means of any simple random number generator, like LCG.
////////////////////////////////////////////////////////////////////////////////
__global__ void RandomGPU(
    float *d_Random,
    int NPerRng,
    unsigned int seed
){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    int iState, iState1, iStateM, iOut;
    unsigned int mti, mti1, mtiM, x;
    unsigned int mt[MT_NN];

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N){
        //Load bit-vector Mersenne Twister parameters
        mt_struct_stripped config = ds_MT[iRng];

        //Initialize current state
        mt[0] = seed;
        for(iState = 1; iState < MT_NN; iState++)
            mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;

        iState = 0;
        mti1 = mt[0];
        for(iOut = 0; iOut < NPerRng; iOut++){
            //iState1 = (iState +     1) % MT_NN
            //iStateM = (iState + MT_MM) % MT_NN
            iState1 = iState + 1;
            iStateM = iState + MT_MM;
            if(iState1 >= MT_NN) iState1 -= MT_NN;
            if(iStateM >= MT_NN) iStateM -= MT_NN;
            mti  = mti1;
            mti1 = mt[iState1];
            mtiM = mt[iStateM];

            x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
            x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? config.matrix_a : 0);
            mt[iState] = x;
            iState = iState1;

            //Tempering transformation
            x ^= (x >> MT_SHIFT0);
            x ^= (x << MT_SHIFTB) & config.mask_b;
            x ^= (x << MT_SHIFTC) & config.mask_c;
            x ^= (x >> MT_SHIFT1);

            //Convert to (0, 1] float and write to global memory
            d_Random[iRng + iOut * MT_RNG_COUNT] = ((float)x + 1.0f) / 4294967296.0f;
        }
    }
}



////////////////////////////////////////////////////////////////////////////////
// Transform each of MT_RNG_COUNT lanes of NPerRng uniformly distributed 
// random samples, produced by RandomGPU(), to normally distributed lanes
// using Cartesian form of Box-Muller transformation.
// NPerRng must be even.
////////////////////////////////////////////////////////////////////////////////
#define PI 3.14159265358979323846264338327950288f

__device__ void BoxMuller(float& u1, float& u2,
                          float& u3, float& u4,
                          float& u5, float& u6,
                          float& u7, float& u8,
                          float& u9, float& u10,
                          float& u11, float& u12,
                          float E0, float Ef,
                          float hdens, float odens,
                          float radius){
    float u,v,w,x,y,z;
    float xtest,ytest,E;
    float hxs,oxs,xs,d,r2,r;
    float costh,sinth,phi;
    float ul,vl,wl,a;
    float sr,ux,vx,wx,vec;
    int i,ie,j,step;
    float myrand[12];

    myrand[0] = u1;
    myrand[1] = u2;
    myrand[2] = u3;
    myrand[3] = u4;
    myrand[4] = u5;
    myrand[5] = u6;
    myrand[6] = u7;
    myrand[7] = u8;
    myrand[8] = u9;
    myrand[9] = u10;
    myrand[10] = u11;
    myrand[11] = u12;

    // hydrogen and oxygen data {{{
    int found = 0;
    float h_xspoint[11];
    float h_epoint[11];
    float h_slope;

    float o_xspoint[84];
    float o_epoint[84];
    float o_slope;

    h_epoint[0] = 0.01;
    h_epoint[1] = 0.1;
    h_epoint[2] = 1.0;
    h_epoint[3] = 1000.0;
    h_epoint[4] = 10000.0;
    h_epoint[5] = 50000.0;
    h_epoint[6] = 1.e5;
    h_epoint[7] = 1.e6;
    h_epoint[8] = 5.e6;
    h_epoint[9] = 1.e7;
    h_epoint[10] = 3.e7;

    h_xspoint[0] = 68.0;
    h_xspoint[1] = 26.0;
    h_xspoint[2] = 20.0;
    h_xspoint[3] = 20.0;
    h_xspoint[4] = 19.0;
    h_xspoint[5] = 15.0;
    h_xspoint[6] = 7.0;
    h_xspoint[7] = 4.0;
    h_xspoint[8] = 1.5;
    h_xspoint[9] = 0.9;
    h_xspoint[10] = 0.3;

    o_xspoint[0] = 3.7;
    o_xspoint[1] = 3.7;
    o_xspoint[2] = 3.7;
    o_xspoint[3] = 3.5;
    o_xspoint[4] = 3.8;
    o_xspoint[5] = 16.0;
    o_xspoint[6] = 4.0;
    o_xspoint[7] = 3.1;
    o_xspoint[8] = 2.7;

    o_xspoint[9] = 8.0;
    o_xspoint[10] = 2.7;
    o_xspoint[11] = 7.0;
    o_xspoint[12] = 2.5;
    o_xspoint[13] = 1.9;
    o_xspoint[14] = 9.0;
    o_xspoint[15] = 1.9;
    o_xspoint[16] = 1.9;
    o_xspoint[17] = 5.0;
    o_xspoint[18] = 1.8;
    o_xspoint[19] = 1.6;
    o_xspoint[20] = 5.0;
    o_xspoint[21] = 2.0;
    o_xspoint[22] = 3.0;
    o_xspoint[23] = 1.5;

    o_xspoint[24] = 1.0;
    o_xspoint[25] = 0.09;
    o_xspoint[26] = 0.7;
    o_xspoint[27] = 1.1;
    o_xspoint[28] = 1.1;
    o_xspoint[29] = 1.4;
    o_xspoint[30] = 5.0;
    o_xspoint[31] = 3.0;
    o_xspoint[32] = 3.0;
    o_xspoint[33] = 5.5;
    o_xspoint[34] = 3.0;
    o_xspoint[35] = 3.0;
    o_xspoint[36] = 5.6;
    o_xspoint[37] = 3.0;
    o_xspoint[38] = 2.0;
    o_xspoint[39] = 1.1;
    o_xspoint[40] = 2.1;

    o_xspoint[41] = 1.4;
    o_xspoint[42] = 2.0;
    o_xspoint[43] = 3.0;
    o_xspoint[44] = 1.3;
    o_xspoint[45] = 2.0;
    o_xspoint[46] = 0.9;
    o_xspoint[47] = 2.0;
    o_xspoint[48] = 0.9;
    o_xspoint[49] = 3.95;
    o_xspoint[50] = 1.4;
    o_xspoint[51] = 1.2;
    o_xspoint[52] = 2.1;
    o_xspoint[53] = 0.8;
    o_xspoint[54] = 1.05;
    o_xspoint[55] = 1.2;
    o_xspoint[56] = 2.6;
    o_xspoint[57] = 1.7;

    o_xspoint[58] = 1.4;
    o_xspoint[59] = 0.8;
    o_xspoint[60] = 1.6;
    o_xspoint[61] = 1.0;
    o_xspoint[62] = 0.8;
    o_xspoint[63] = 1.7;
    o_xspoint[64] = 0.6;
    o_xspoint[65] = 1.1;
    o_xspoint[66] = 1.1;
    o_xspoint[67] = 1.5;
    o_xspoint[68] = 0.85;
    o_xspoint[69] = 0.85;
    o_xspoint[70] = 1.9;
    o_xspoint[71] = 1.4;
    o_xspoint[72] = 1.1;

    o_xspoint[73] = 1.5;
    o_xspoint[74] = 0.8;
    o_xspoint[75] = 1.4;
    o_xspoint[76] = 1.1;
    o_xspoint[77] = 1.5;
    o_xspoint[78] = 1.1;
    o_xspoint[79] = 1.35;
    o_xspoint[80] = 1.08;
    o_xspoint[81] = 1.4;
    o_xspoint[82] = 1.2;
    o_xspoint[83] = 1.6;

    o_epoint[0] = 0.01;
    o_epoint[1] = 0.1;
    o_epoint[2] = 1.e4;
    o_epoint[3] = 3.e5;
    o_epoint[4] = 3.5e5;
    o_epoint[5] = 4.4e5;
    o_epoint[6] = 5.2e5;
    o_epoint[7] = 6.e5;

    o_epoint[8] = 8.5e5;
    o_epoint[9] = 1.e6;
    o_epoint[10] = 1.2e6;
    o_epoint[11] = 1.31e6;
    o_epoint[12] = 1.4e6;
    o_epoint[13] = 1.64e6;
    o_epoint[14] = 1.65e6;

    o_epoint[15] = 1.66e6;
    o_epoint[16] = 1.67e6;
    o_epoint[17] = 1.68e6;
    o_epoint[18] = 1.69e6;

    o_epoint[19] = 1.8e6;
    o_epoint[20] = 1.82e6;
    o_epoint[21] = 1.83e6;
    o_epoint[22] = 1.9e6;
    o_epoint[23] = 2.0e6;
    o_epoint[24] = 2.25e6;
    o_epoint[25] = 2.35e6;

    o_epoint[26] = 2.4e6;
    o_epoint[27] = 2.5e6;
    o_epoint[28] = 2.9e6;
    o_epoint[29] = 3.1e6;

    o_epoint[30] = 3.2e6;
    o_epoint[31] = 3.21e6;
    o_epoint[32] = 3.4e6;
    o_epoint[33] = 3.41e6;
    o_epoint[34] = 3.42e6;
    o_epoint[35] = 3.78e6;
    o_epoint[36] = 3.79e6;
    o_epoint[37] = 3.8e6;

    o_epoint[38] = 4.e6;
    o_epoint[39] = 4.1e6;
    o_epoint[40] = 4.4e6;
    o_epoint[41] = 4.45e6;
    o_epoint[42] = 4.5e6;
    o_epoint[43] = 4.51e6;

    o_epoint[44] = 4.52e6;
    o_epoint[45] = 4.6e6;
    o_epoint[46] = 4.7e6;
    o_epoint[47] = 4.81e6;
    o_epoint[48] = 5.e6;
    o_epoint[49] = 5.15e6;
    o_epoint[50] = 5.2e6;

    o_epoint[51] = 5.35e6;
    o_epoint[52] = 5.36e6;
    o_epoint[53] = 5.37e6;
    o_epoint[54] = 5.4e6;
    o_epoint[55] = 5.65e6;
    o_epoint[56] = 5.66e6;
    o_epoint[57] = 5.67e6;

    o_epoint[58] = 5.98e6;
    o_epoint[59] = 5.99e6;

    o_epoint[60] = 6.05e6;
    o_epoint[61] = 6.06e6;
    o_epoint[62] = 6.3e6;
    o_epoint[63] = 6.4e6;
    o_epoint[64] = 6.5e6;
    o_epoint[65] = 6.8e6;
    o_epoint[66] = 6.85e6;

    o_epoint[67] = 6.86e6;
    o_epoint[68] = 6.88e6;
    o_epoint[69] = 7.e6;
    o_epoint[70] = 7.2e6;
    o_epoint[71] = 7.4e6;
    o_epoint[72] = 7.5e6;
    o_epoint[73] = 7.8e6;

    o_epoint[74] = 8.05e6;
    o_epoint[75] = 8.3e6;
    o_epoint[76] = 8.6e6;
    o_epoint[77] = 8.7e6;
    o_epoint[78] = 9.e6;
    o_epoint[79] = 9.1e6;
    o_epoint[80] = 9.3e6;

    o_epoint[81] = 11.e6;
    o_epoint[82] = 12.e6;
    o_epoint[83] = 18.e6;
    // }}}

    u = 0.0;
    v = 0.0;
    w = 1.0;

    // source location
    x = 0.0;
    y = 0.0;
    z = 0.0;

    step = 0;
    for (j=0;j<11;j++) {
      switch(step)
      {
        case 0: 
          xtest = -logf(myrand[j]);  
          step = 1;
          break;
        case 1: 
          ytest = -logf(myrand[j]);  

          if ( powf(ytest-xtest-1.0,2) <= 4.*xtest ) {
            E = 2.*xtest*1.e6; // we are working in eV
            step = 2;
          } else {
            step = 0;
          } 
          break;
        case 2: 
          // hydrogen {{{
          found = 0;
          for (i=1;i<11;i++) {
            ie = i;
            if ( E < h_epoint[ie] ) {
              found = 1;
              break;
            }
          }
          if ( found == 0 ) {
            // E >= 3E7
            hxs = hdens*h_xspoint[ie];
          } else {
            h_slope = (h_xspoint[ie] - h_xspoint[ie-1])/logf(h_epoint[ie]/h_epoint[ie-1]);
            hxs = h_slope*logf(E/h_epoint[ie-1])+h_xspoint[ie-1]; // extrapolate between ie-1 and ie
            hxs = hxs*hdens;
          }
          // }}}

          // oxygen {{{
          found = 0;
          for (i=1;i<84;i++) {
            ie = i;
            if ( E < o_epoint[ie] ) {
              found = 1;
              break;
            }
          }
          if ( found == 0 ) {
            // E >= 1.8E7
            oxs = odens*o_xspoint[ie];
          } else {
            o_slope = (o_xspoint[ie] - o_xspoint[ie-1])/logf(o_epoint[ie]/o_epoint[ie-1]);
            oxs = o_slope*logf(E/o_epoint[ie-1])+o_xspoint[ie-1]; // extrapolate between ie-1 and ie
            oxs = oxs*odens;
          }
          // }}}

          // macro cross section xs in 1/cm
          xs = hxs + oxs;

          // get a flight path d in cm
          d = -logf(myrand[j])/xs;

          // collision site
          x = x + d*u;
          y = y + d*v;
          z = z + d*w;

          // radius of collision site
          r2 = x*x + y*y + z*z;
          r = sqrtf(r2);

          if ( r >= radius ) {
            // the particle escaped
            step = 0;

            u = 0.0;
            v = 0.0;
            w = 1.0;

            // source location
            x = 0.0;
            y = 0.0;
            z = 0.0;
          } else {
            step = 3;
          } 
          break;
        case 3: 
          // random number to select type of atom
          if (myrand[j] < hxs/xs) {
            // if collision in hydrogen
            a = 1.0;
          } else {
            // if collision in oxygen
            a = 16.0;
          }
          step = 4;
          break;
        case 4: 
          costh = 2.*myrand[j] - 1.0;
          step = 5;
          break;
        case 5: 
          phi = 2.*myrand[j]*PI;
          step = 6;

          wl = (a*costh+1.0)/sqrtf(a*a+2.*a*costh+1.0);
          sinth = sqrtf(1.-wl*wl);

          ul = sinth*cosf(phi);
          vl = sinth*sinf(phi);
 
          // post-collision energy
          E = E*(a*a + 2.*a*costh+1.0)/powf(a+1.0,2);

          if ( fabs(u) < 0.9 ) {
            // x-axis transformation Eqn. 4.43
            sr = sqrtf(1.0-u*u);
            ux = sr*ul + u*wl;
            vx = -u*v*ul/sr + w*vl/sr + v*wl;
            wx = -w*u*ul/sr - v*vl/sr + w*wl;
          } else {
            // y-axis transformation Eqn. 4.44
            sr = sqrt(1.0-v*v);
            ux = w*ul/sr - u*v*vl/sr + u*wl;
            vx = vl*sr + v*wl;
            wx = -u*ul/sr - v*w*vl/sr + w*wl;
          }
          vec = sqrtf(ux*ux + vx*vx + wx*wx);
               
          // normalized lab direction cosines
          u = ux/vec;
          v = vx/vec;
          w = wx/vec;

          if ( E > Ef ) {
            step = 2;
          } else {
            step = 6;
          }
          break;
      }
      if ( step == 6 ) break;
    }

    if ( step == 6 ) {
      u1 = r2;
    } else {
      u1 = -1.0;
    }
    u2 = 15.0;
    u3 = -1.0;
    u4 = -1.0;
    u5 = -1.0;
    u6 = -1.0;
    u7 = -1.0;
    u8 = -1.0;
    u9 = -1.0;
    u10 = -1.0;
    u11 = -1.0;
    u12 = -1.0;
}

__global__ void BoxMullerGPU(float *d_Random, int NPerRng,
                             float E0,float Ef,float hdens,
                             float odens, float radius){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N)
        for(int iOut = 0; iOut < NPerRng; iOut += 12)
            BoxMuller(
                d_Random[iRng + (iOut + 0) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 1) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 2) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 3) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 4) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 5) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 6) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 7) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 8) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 9) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 10) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 11) * MT_RNG_COUNT],
                E0,Ef,hdens,odens,radius
            );
}
